#include "hip/hip_runtime.h"
#include <glm/gtx/norm.hpp>

#include <kernels/sphGPU.h>
#include <neighborTable.h>
#include <sph.h>
#include <timer.h>

/// Returns a hash of the cell position
__device__ uint16_t getHashDevice(const glm::ivec3 &cell)
{
    return (
        (uint)(cell.x * 73856093)
        ^ (uint)(cell.y * 19349663)
        ^ (uint)(cell.z * 83492791)
    ) % TABLE_SIZE;
}

/// Get the cell that the particle is in.
__device__ glm::ivec3 getCellDevice(Particle *p, float h)
{
    return {p->position.x / h, p->position.y / h, p->position.z / h};
}

///// Calculate particle hashes
//__global__ void calculateHashesKernel(
//    Particle *particles, const size_t particleCount, float h)
//{
//    size_t particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
//    if (particleIndex >= particleCount) {
//        return;
//    }
//    Particle *particle = &particles[particleIndex];
//    particles->hash = getHashDevice(getCellDevice(particle, h));
//}
//
//struct HashComp
//{
//    __host__ __device__ bool operator()(
//            const Particle& p1, const Particle& p2) {
//        return p1.hash < p2.hash;
//    }
//};

///// Sort particles by hash
//__global__ void sortParticles(Particle *particles, const size_t particleCount)
//{
//    thrust::sort(particles, particles + particleCount, HashComp());
//}
//
///// Constructs the neighbor table and stores the result in `createdTable`.
//__global__ void constructNeighborTable(
//    Particle *sortedParticles, const size_t particleCount,
//    uint32_t **createdTable)
//{
//    size_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
//    if (globalThreadIdx >= 1) {
//        // This method should only be run on one thread. This is here
//        // for safety.
//        return;
//    }
//
//    uint32_t *particleTable
//        = (uint32_t *)malloc(sizeof(uint32_t) * TABLE_SIZE);
//    for (size_t i = 0; i < TABLE_SIZE; ++i) {
//        particleTable[i] = NO_PARTICLE;
//    }
//
//    uint32_t prevHash = NO_PARTICLE;
//    for (size_t i = 0; i < particleCount; ++i) {
//        uint16_t currentHash = sortedParticles[i].hash;
//        if (currentHash != prevHash) {
//            particleTable[currentHash] = i;
//            prevHash = currentHash;
//        }
//    }
//
//    createdTable = &particleTable;
//}

/// Kernel computation function for calculating density
/// and pressures of particles in the given SPH System.
__global__ void calculateDensitiesAndPressuresKernel(
    Particle *particles, const size_t particleCount,
    const uint32_t *particleTable, const SPHSettings settings)
{
    size_t piIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (piIndex > particleCount) {
        return;
    }
    Particle *pi = &particles[piIndex];

    // TODO: Try make use of CUDA constants
    float massPoly6Product = settings.mass * settings.poly6;
    glm::ivec3 cell = getCellDevice(pi, settings.h);

    float pDensity = 0;

    for (int x = -1; x <= 1; x++) {
        for (int y = -1; y <= 1; y++) {
            for (int z = -1; z <= 1; z++) {
                uint16_t cellHash
                    = getHashDevice(cell + glm::ivec3(x, y, z));
                uint32_t pjIndex = particleTable[cellHash];
                if (pjIndex == NO_PARTICLE) {
                    continue;
                }
                while (pjIndex < particleCount) {
                    if (pjIndex == piIndex) {
                        pjIndex++;
                        continue;
                    }
                    Particle *pj = &particles[pjIndex];
                    if (pj->hash != cellHash) {
                        break;
                    }
                    float dist2 = glm::length2(pj->position - pi->position);
                    if (dist2 < settings.h2 && pi != pj) {
                        pDensity += massPoly6Product
                            * glm::pow(settings.h2 - dist2, 3);
                    }
                    pjIndex++;
                }
            }
        }
    }

    // Include self density (as itself isn't included in neighbour)
    pi->density = pDensity + settings.selfDens;

    // Calculate pressure
    float pPressure
        = settings.gasConstant * (pi->density - settings.restDensity);
    pi->pressure = pPressure;
}

/// Parallel computation function for calculating forces
/// of particles in the given SPH System.
__global__ void calculateForcesKernel(
    Particle *particles, const size_t particleCount,
    const uint32_t *particleTable, const SPHSettings settings)
{
    size_t piIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (piIndex > particleCount) {
        return;
    }
    Particle *pi = &particles[piIndex];

    // Another constant
    glm::ivec3 cell = getCellDevice(pi, settings.h);

    pi->force = glm::vec3(0);

    for (int x = -1; x <= 1; x++) {
        for (int y = -1; y <= 1; y++) {
            for (int z = -1; z <= 1; z++) {
                uint16_t cellHash
                    = getHashDevice(cell + glm::ivec3(x, y, z));
                uint32_t pjIndex = particleTable[cellHash];
                if (pjIndex == NO_PARTICLE) {
                    continue;
                }
                while (pjIndex < particleCount) {
                    if (pjIndex == piIndex) {
                        pjIndex++;
                        continue;
                    }
                    Particle *pj = &particles[pjIndex];
                    if (pj->hash != cellHash) {
                        break;
                    }
                    float dist2 = glm::length2(pj->position - pi->position);
                    if (dist2 < settings.h2 && pi != pj) {
                        //unit direction and length
                        float dist = sqrt(dist2);
                        glm::vec3 dir = glm::normalize(pj->position - pi->position);

                        //apply pressure force
                        glm::vec3 pressureForce = -dir * settings.mass * (pi->pressure + pj->pressure) / (2 * pj->density) * settings.spikyGrad;
                        pressureForce *= std::pow(settings.h - dist, 2);
                        pi->force += pressureForce;

                        //apply viscosity force
                        glm::vec3 velocityDif = pj->velocity - pi->velocity;
                        glm::vec3 viscoForce = settings.viscosity * settings.mass * (velocityDif / pj->density) * settings.spikyLap * (settings.h - dist);
                        pi->force += viscoForce;
                    }
                    pjIndex++;
                }
            }
        }
	}
}

/// Parallel computation function moving positions
/// of particles in the given SPH System.
__global__ void updateParticlePositionsKernel(
    Particle *particles, const size_t particleCount,
    glm::mat4 *particleTransforms, const SPHSettings settings,
    const float deltaTime)
{
    size_t pIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (pIndex > particleCount) {
        return;
    }
    Particle *p = &particles[pIndex];

    // TODO: These should be constants somewhere else.
    glm::mat4 sphereScale = glm::scale(glm::vec3(settings.h / 2.f));
    float boxWidth = 3.f;
    float elasticity = 0.5f;

    //calculate acceleration and velocity
    glm::vec3 acceleration = p->force / p->density + glm::vec3(0, settings.g, 0);
    p->velocity += acceleration * deltaTime;

    // Update position
    p->position += p->velocity * deltaTime;

    // Handle collisions with box
    if (p->position.y < settings.h) {
        p->position.y = -p->position.y + 2 * settings.h + 0.0001f;
        p->velocity.y = -p->velocity.y * elasticity;
    }

    if (p->position.x < settings.h - boxWidth) {
        p->position.x = -p->position.x + 2 * (settings.h - boxWidth) + 0.0001f;
        p->velocity.x = -p->velocity.x * elasticity;
    }

    if (p->position.x > -settings.h + boxWidth) {
        p->position.x = -p->position.x + 2 * -(settings.h - boxWidth) - 0.0001f;
        p->velocity.x = -p->velocity.x * elasticity;
    }

    if (p->position.z < settings.h - boxWidth) {
        p->position.z = -p->position.z + 2 * (settings.h - boxWidth) + 0.0001f;
        p->velocity.z = -p->velocity.z * elasticity;
    }

    if (p->position.z > -settings.h + boxWidth) {
        p->position.z = -p->position.z + 2 * -(settings.h - boxWidth) - 0.0001f;
        p->velocity.z = -p->velocity.z * elasticity;
    }

    particleTransforms[pIndex] = glm::translate(p->position) * sphereScale;
}

void updateParticlesGPU(
    Particle *particles, glm::mat4 *particleTransforms,
    const size_t particleCount, const SPHSettings &settings,
    float deltaTime)
{
//    std::cout << "Running on GPU" << std::endl;

    const size_t threadCount = std::thread::hardware_concurrency();
    std::thread threads[threadCount];

    size_t blockBoundaries[threadCount + 1];
    blockBoundaries[0] = 0;
    size_t blockSize = particleCount / threadCount;
    for (size_t i = 1; i < threadCount; i++) {
        blockBoundaries[i] = i * blockSize;
    }
    blockBoundaries[threadCount] = particleCount;

    // Calculate hashes
    {
        Timer timer("hashes");
        for (int i = 0; i < threadCount; i++) {
            threads[i] = std::thread(
                parallelCalculateHashes, particles, blockBoundaries[i],
                blockBoundaries[i + 1], settings);
        }
        for (std::thread& thread : threads) {
            thread.join();
        }
    }

    // Sort particles
    {
        Timer timer("sort");
        sortParticles(particles, particleCount);
    }

    // Copy particles
    Particle *dParticles;
    size_t particlesSize = sizeof(Particle) * particleCount;
    hipMalloc((void**)&dParticles, particlesSize);
    hipMemcpy(dParticles, particles, particlesSize, hipMemcpyHostToDevice);

    // Copy particle transforms
    glm::mat4 *dParticleTransforms;
    size_t transformsSize = sizeof(glm::mat4) * particleCount;
    hipMalloc((void**)&dParticleTransforms, transformsSize);
    hipMemcpy(
        dParticleTransforms, particleTransforms, transformsSize,
        hipMemcpyHostToDevice);

    // Create and copy particle table
    uint32_t *dParticleTable;
    {
        Timer timer("tableCreation");
        uint32_t *particleTable
            = createNeighborTable(particles, particleCount);
        size_t tableSize = sizeof(uint32_t) * TABLE_SIZE;
        hipMalloc((void**)&dParticleTable, tableSize);
        hipMemcpy(
            dParticleTable, particleTable, tableSize, hipMemcpyHostToDevice);
        // Can free the host table since it won't be of use.
        free(particleTable);
    }

//    const size_t blockSize = 512;
//    size_t gridSize = blockSize / particleCount + 1;
//    calculateHashesKernel<<<gridSize, blockSize>>>(
//        dParticles, particleCount, settings.h);
//    hipDeviceSynchronize();
//
//    uint32_t *dParticleTable;
//    constructNeighborTable<<<1, 1>>>(
//        particles, particleCount, &dParticleTable);
//    hipDeviceSynchronize();

    size_t threadsPerBlock = 512;
    size_t gridSize = particleCount / threadsPerBlock + 1;

    {
        Timer timer("densities");
        calculateDensitiesAndPressuresKernel<<<gridSize, threadsPerBlock>>>(
            dParticles, particleCount, dParticleTable, settings);
        hipDeviceSynchronize();
    }
//    // Check for errors
//    hipError_t error = hipGetLastError();
//    if (error != hipSuccess) {
//        printf("CUDA error: %s\n", hipGetErrorString(error));
//    }

    {
        Timer timer("forces");
        calculateForcesKernel<<<gridSize, threadsPerBlock>>>(
            dParticles, particleCount, dParticleTable, settings);
        hipDeviceSynchronize();
    }

    {
        Timer timer("positions");
        updateParticlePositionsKernel<<<gridSize, threadsPerBlock>>>(
            dParticles, particleCount, dParticleTransforms, settings, deltaTime);
        hipDeviceSynchronize();
    }

    hipMemcpy(particles, dParticles, particlesSize, hipMemcpyDeviceToHost);
    hipMemcpy(
        particleTransforms, dParticleTransforms, transformsSize,
        hipMemcpyDeviceToHost);

    // Free allocated memory
    hipFree(dParticles);
    hipFree(dParticleTransforms);
    hipFree(dParticleTable);
}