
#include <hip/hip_runtime.h>
//#include <glm/gtx/norm.hpp>
//
//#include <kernels/sphGPU.h>
//#include <neighborTable.h>
//
///// Main kernel for running SPH and updating particle positions
//__global__ void updateParticlesKernel(
//    Particle *particles, glm::mat4 *particleTransforms,
//    const size_t particleCount, const SPHSettings &settings,
//    float deltaTime)
//{
//    size_t blockWidth = particleCount / blockDim.x + 1;
//    size_t blockOffset = threadIdx.x * blockWidth;
//    size_t maybeEnd = blockWidth + blockOffset;
//    size_t end = maybeEnd < particleCount ? maybeEnd : particleCount;
//    for (size_t i = blockOffset; i < end; i++) {
//        particleTransforms[i] *= glm::translate(glm::vec3(0, deltaTime * 10, 0));
//    }
//}
//
///// Returns a hash of the cell position
//__device__ uint getHashDevice(const glm::ivec3 &cell)
//{
//    return (
//        (uint)(cell.x * 73856093)
//        ^ (uint)(cell.y * 19349663)
//        ^ (uint)(cell.z * 83492791)
//    ) % TABLE_SIZE;
//}
//
///// Get the cell that the particle is in.
//__device__ glm::ivec3 getCellDevice(Particle *p, float h)
//{
//    return {p->position.x / h, p->position.y / h, p->position.z / h};
//}
//
///// Kernel computation function for calculating density
///// and pressures of particles in the given SPH System.
//__global__ void calculateDensitiesAndPressuresKernel(
//    Particle *particles, const size_t particleCount, const int *particleTable,
//    const SPHSettings settings)
//{
//    // TODO: Try make use of CUDA constants
//	float massPoly6Product = settings.mass * settings.poly6;
//
//    // TODO: Try make use of CUDA constants
//    size_t blockWidth = particleCount / (blockDim.x * gridDim.x) + 1;
//    size_t start = (blockIdx.x * blockDim.x + threadIdx.x) * blockWidth;
//    size_t maybeEnd = start + blockWidth;
//    size_t end = maybeEnd < particleCount ? maybeEnd : particleCount;
//
//    for (size_t i = start; i < end; i++) {
//		float pDensity = 0;
//        Particle* pi = &particles[i];
//        glm::ivec3 cell = getCellDevice(pi, settings.h);
//
//		for (int x = -1; x <= 1; x++) {
//			for (int y = -1; y <= 1; y++) {
//				for (int z = -1; z <= 1; z++) {
//					glm::ivec3 near_cell = cell + glm::ivec3(x, y, z);
//					uint index = getHashDevice(near_cell);
//
//                    // Iterate through cell linked list
//                    int pjIndex = particleTable[index];
//                    while (pjIndex != -1) {
//                        Particle *pj = &particles[pjIndex];
//                        float dist2 = glm::length2(pj->position - pi->position);
//                        if (dist2 < settings.h2 && pi != pj) {
//                            pDensity += massPoly6Product
//                                * glm::pow(settings.h2 - dist2, 3);
//                        }
//                        pjIndex = pj->next;
//                    }
//				}
//			}
//		}
//
//		// Include self density (as itself isn't included in neighbour)
//		pi->density = pDensity + settings.selfDens;
//
//		// Calculate pressure
//		float pPressure
//            = settings.gasConstant * (pi->density - settings.restDensity);
//		pi->pressure = pPressure;
//	}
//}
//
///// Parallel computation function for calculating forces
///// of particles in the given SPH System.
//__global__ void calculateForcesKernel(
//    Particle *particles, const size_t particleCount, const int *particleTable,
//    const SPHSettings settings)
//{
//	// TODO: Try make use of CUDA constants
//    size_t blockWidth = particleCount / (blockDim.x * gridDim.x) + 1;
//    size_t start = (blockIdx.x * blockDim.x + threadIdx.x) * blockWidth;
//    size_t maybeEnd = start + blockWidth;
//    size_t end = maybeEnd < particleCount ? maybeEnd : particleCount;
//
//    for (size_t i = start; i < end; i++) {
//		Particle* pi = &particles[i];
//		pi->force = glm::vec3(0);
//		glm::ivec3 cell = getCellDevice(pi, settings.h);
//
//		for (int x = -1; x <= 1; x++) {
//			for (int y = -1; y <= 1; y++) {
//				for (int z = -1; z <= 1; z++) {
//					glm::ivec3 near_cell = cell + glm::ivec3(x, y, z);
//					uint index = getHashDevice(near_cell);
//                    int pjIndex = particleTable[index];
//
//					// Iterate through cell linked list
//					while (pjIndex != -1) {
//                        Particle *pj = &particles[pjIndex];
//						float dist2 = glm::length2(pj->position - pi->position);
//						if (dist2 < settings.h2 && pi != pj) {
//							//unit direction and length
//							float dist = sqrt(dist2);
//							glm::vec3 dir = glm::normalize(pj->position - pi->position);
//
//							//apply pressure force
//							glm::vec3 pressureForce = -dir * settings.mass * (pi->pressure + pj->pressure) / (2 * pj->density) * settings.spikyGrad;
//							pressureForce *= std::pow(settings.h - dist, 2);
//							pi->force += pressureForce;
//
//							//apply viscosity force
//							glm::vec3 velocityDif = pj->velocity - pi->velocity;
//							glm::vec3 viscoForce = settings.viscosity * settings.mass * (velocityDif / pj->density) * settings.spikyLap * (settings.h - dist);
//							pi->force += viscoForce;
//						}
//                        pjIndex = pj->next;
//					}
//				}
//			}
//		}
//	}
//}
//
///// Parallel computation function moving positions
///// of particles in the given SPH System.
//__global__ void updateParticlePositionsKernel(
//    Particle *particles, const size_t particleCount,
//    glm::mat4 *particleTransforms, const SPHSettings settings,
//    const float deltaTime)
//{
//    glm::mat4 sphereScale = glm::scale(glm::vec3(settings.h / 2.f));
//    float boxWidth = 3.f;
//    float elasticity = 0.5f;
//
//    // TODO: Try make use of CUDA constants
//    size_t blockWidth = particleCount / (blockDim.x * gridDim.x) + 1;
//    size_t start = (blockIdx.x * blockDim.x + threadIdx.x) * blockWidth;
//    size_t maybeEnd = start + blockWidth;
//    size_t end = maybeEnd < particleCount ? maybeEnd : particleCount;
//
//    for (size_t i = start; i < end; i++) {
//		Particle *p = &particles[i];
//
//		//calculate acceleration and velocity
//		glm::vec3 acceleration = p->force / p->density + glm::vec3(0, settings.g, 0);
//		p->velocity += acceleration * deltaTime;
//
//		// Update position
//		p->position += p->velocity * deltaTime;
//
//		// Handle collisions with box
//		if (p->position.y < settings.h) {
//			p->position.y = -p->position.y + 2 * settings.h + 0.0001f;
//			p->velocity.y = -p->velocity.y * elasticity;
//		}
//
//		if (p->position.x < settings.h - boxWidth) {
//			p->position.x = -p->position.x + 2 * (settings.h - boxWidth) + 0.0001f;
//			p->velocity.x = -p->velocity.x * elasticity;
//		}
//
//		if (p->position.x > -settings.h + boxWidth) {
//			p->position.x = -p->position.x + 2 * -(settings.h - boxWidth) - 0.0001f;
//			p->velocity.x = -p->velocity.x * elasticity;
//		}
//
//		if (p->position.z < settings.h - boxWidth) {
//			p->position.z = -p->position.z + 2 * (settings.h - boxWidth) + 0.0001f;
//			p->velocity.z = -p->velocity.z * elasticity;
//		}
//
//		if (p->position.z > -settings.h + boxWidth) {
//			p->position.z = -p->position.z + 2 * -(settings.h - boxWidth) - 0.0001f;
//			p->velocity.z = -p->velocity.z * elasticity;
//		}
//
//        particleTransforms[i] = glm::translate(p->position) * sphereScale;
//	}
//}
//
//void updateParticlesGPU(
//    Particle *particles, glm::mat4 *particleTransforms,
//    const size_t particleCount, const SPHSettings &settings,
//    float deltaTime)
//{
//    std::cout << "Running on GPU" << std::endl;
//    // Create the neighbor table first as it adjusts some of the
//    // particle properties
//    int *neighborTable
//        = createNeighborTable(particles, particleCount, settings);
//
//    // Copy particles
//    Particle *dParticles;
//    size_t particlesSize = sizeof(Particle) * particleCount;
//    cudaMalloc((void**)&dParticles, particlesSize);
//    cudaMemcpy(dParticles, particles, particlesSize, cudaMemcpyHostToDevice);
//
//    // Copy particle transforms
//    glm::mat4 *dParticleTransforms;
//    size_t transformsSize = sizeof(glm::mat4) * particleCount;
//    cudaMalloc((void**)&dParticleTransforms, transformsSize);
//    cudaMemcpy(
//        dParticleTransforms, particleTransforms, transformsSize,
//        cudaMemcpyHostToDevice);
//
//    // Copy neighbor table
//    int *dParticleTable;
//    size_t particleTableSize = sizeof(int) * TABLE_SIZE;
//    cudaMalloc((void**)&dParticleTable, particleTableSize);
//    cudaMemcpy(
//        dParticleTable, neighborTable, particleTableSize,
//        cudaMemcpyHostToDevice);
//    free(neighborTable);
//
//    calculateDensitiesAndPressuresKernel<<<8, 512>>>(
//        dParticles, particleCount, dParticleTable, settings);
//    cudaDeviceSynchronize();
////    // Check for errors
////    cudaError_t error = cudaGetLastError();
////    if (error != cudaSuccess) {
////        printf("CUDA error: %s\n", cudaGetErrorString(error));
////    }
//
//    calculateForcesKernel<<<8, 512>>>(
//        dParticles, particleCount, dParticleTable, settings);
//    cudaDeviceSynchronize();
//
//    updateParticlePositionsKernel<<<8, 512>>>(
//        dParticles, particleCount, dParticleTransforms, settings, deltaTime);
//    cudaDeviceSynchronize();
//
//    cudaMemcpy(particles, dParticles, particlesSize, cudaMemcpyDeviceToHost);
//    cudaMemcpy(
//        particleTransforms, dParticleTransforms, transformsSize,
//        cudaMemcpyDeviceToHost);
//
//    // Free allocated memory
//    cudaFree(dParticles);
//    cudaFree(dParticleTransforms);
//    cudaFree(dParticleTable);
//}